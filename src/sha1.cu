#include "hip/hip_runtime.h"
/**
 * @private
 * @headerfile sha1.cuh <sha1.cuh>
 * @copyright This file is released into the Public Domain under
 * the Creative Commons Zero v1.0 Universal license.
*/

/* include guard */
#ifndef CRYPTO_SHA1_CU
#define CRYPTO_SHA1_CU


#include "sha1.cuh"
#include <string.h>

/**
 * @private
 * SHA1 transformation rounds.
 * @param ctx Pointer to SHA1 context
 * @param data Pointer to input to be transformed
*/
__device__ void cu_sha1_transform(SHA1_CTX *ctx, const uint8_t data[])
{
   /**
    * @private
    * SHA1 transformation constant.
   */
   __device__ __constant__ __align__(32) static uint32_t k[4] = {
      0x5a827999, 0x6ed9eba1, 0x8f1bbcdc, 0xca62c1d6
   };

   __align__(8) uint32_t W[16];
   uint32_t a, b, c, d, e;

   /* copy data into intermediate state */
   memcpy(W, data, 64);

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];

	/* SHA1 round 1 */
   sha1_r0(a, b, c, d, e, 0);
   sha1_r0(e, a, b, c, d, 1);
   sha1_r0(d, e, a, b, c, 2);
   sha1_r0(c, d, e, a, b, 3);
   sha1_r0(b, c, d, e, a, 4);
   sha1_r0(a, b, c, d, e, 5);
   sha1_r0(e, a, b, c, d, 6);
   sha1_r0(d, e, a, b, c, 7);
   sha1_r0(c, d, e, a, b, 8);
   sha1_r0(b, c, d, e, a, 9);
   sha1_r0(a, b, c, d, e, 10);
   sha1_r0(e, a, b, c, d, 11);
   sha1_r0(d, e, a, b, c, 12);
   sha1_r0(c, d, e, a, b, 13);
   sha1_r0(b, c, d, e, a, 14);
   sha1_r0(a, b, c, d, e, 15);
   /* alternate round computation */
   sha1_r1(e, a, b, c, d, 16);
   sha1_r1(d, e, a, b, c, 17);
   sha1_r1(c, d, e, a, b, 18);
   sha1_r1(b, c, d, e, a, 19);
   sha1_r2(a, b, c, d, e, 20);

	/* SHA1 round 2 */
   sha1_r2(e, a, b, c, d, 21);
   sha1_r2(d, e, a, b, c, 22);
   sha1_r2(c, d, e, a, b, 23);
   sha1_r2(b, c, d, e, a, 24);
   sha1_r2(a, b, c, d, e, 25);
   sha1_r2(e, a, b, c, d, 26);
   sha1_r2(d, e, a, b, c, 27);
   sha1_r2(c, d, e, a, b, 28);
   sha1_r2(b, c, d, e, a, 29);
   sha1_r2(a, b, c, d, e, 30);
   sha1_r2(e, a, b, c, d, 31);
   sha1_r2(d, e, a, b, c, 32);
   sha1_r2(c, d, e, a, b, 33);
   sha1_r2(b, c, d, e, a, 34);
   sha1_r2(a, b, c, d, e, 35);
   sha1_r2(e, a, b, c, d, 36);
   sha1_r2(d, e, a, b, c, 37);
   sha1_r2(c, d, e, a, b, 38);
   sha1_r2(b, c, d, e, a, 39);

	/* SHA1 round 3 */
   sha1_r3(a, b, c, d, e, 40);
   sha1_r3(e, a, b, c, d, 41);
   sha1_r3(d, e, a, b, c, 42);
   sha1_r3(c, d, e, a, b, 43);
   sha1_r3(b, c, d, e, a, 44);
   sha1_r3(a, b, c, d, e, 45);
   sha1_r3(e, a, b, c, d, 46);
   sha1_r3(d, e, a, b, c, 47);
   sha1_r3(c, d, e, a, b, 48);
   sha1_r3(b, c, d, e, a, 49);
   sha1_r3(a, b, c, d, e, 50);
   sha1_r3(e, a, b, c, d, 51);
   sha1_r3(d, e, a, b, c, 52);
   sha1_r3(c, d, e, a, b, 53);
   sha1_r3(b, c, d, e, a, 54);
   sha1_r3(a, b, c, d, e, 55);
   sha1_r3(e, a, b, c, d, 56);
   sha1_r3(d, e, a, b, c, 57);
   sha1_r3(c, d, e, a, b, 58);
   sha1_r3(b, c, d, e, a, 59);

	/* SHA1 round 4 */
   sha1_r4(a, b, c, d, e, 60);
   sha1_r4(e, a, b, c, d, 61);
   sha1_r4(d, e, a, b, c, 62);
   sha1_r4(c, d, e, a, b, 63);
   sha1_r4(b, c, d, e, a, 64);
   sha1_r4(a, b, c, d, e, 65);
   sha1_r4(e, a, b, c, d, 66);
   sha1_r4(d, e, a, b, c, 67);
   sha1_r4(c, d, e, a, b, 68);
   sha1_r4(b, c, d, e, a, 69);
   sha1_r4(a, b, c, d, e, 70);
   sha1_r4(e, a, b, c, d, 71);
   sha1_r4(d, e, a, b, c, 72);
   sha1_r4(c, d, e, a, b, 73);
   sha1_r4(b, c, d, e, a, 74);
   sha1_r4(a, b, c, d, e, 75);
   sha1_r4(e, a, b, c, d, 76);
   sha1_r4(d, e, a, b, c, 77);
   sha1_r4(c, d, e, a, b, 78);
   sha1_r4(b, c, d, e, a, 79);

	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
}  /* end cu_sha1_transform() */

/**
 * Initialize a SHA1 context.
 * @param ctx Pointer to SHA1 context
*/
__device__ void cu_sha1_init(SHA1_CTX *ctx)
{
   ctx->datalen = 0;
   ctx->bitlen[0] = ctx->bitlen[1] = 0;
   ctx->state[0] = 0x67452301;
   ctx->state[1] = 0xEFCDAB89;
   ctx->state[2] = 0x98BADCFE;
   ctx->state[3] = 0x10325476;
   ctx->state[4] = 0xc3d2e1f0;
}  /* end cu_sha1_init() */

/**
 * Add @a inlen bytes from @a in to a SHA1 context for hashing.
 * @param ctx Pointer to SHA1 context
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
*/
__device__ void cu_sha1_update(SHA1_CTX *ctx, const void *in, size_t inlen)
{
   size_t i, n;
   uint32_t old;

   for(i = n = 0; inlen; i += n, inlen -= n) {
      /* copy memory to input buffer in chunks */
      n = (ctx->datalen + inlen) > 64 ? 64 - ctx->datalen : inlen;
      memcpy(ctx->data + ctx->datalen, (const uint8_t *) in + i, n);
      ctx->datalen += n;
      /* process input buffer */
      if (ctx->datalen == 64) {
         cu_sha1_transform(ctx, ctx->data);
         ctx->datalen = 0;
         old = ctx->bitlen[0];
         ctx->bitlen[0] += 512;
         if (ctx->bitlen[0] < old) ctx->bitlen[1]++;  /* add in carry */
      }
   }
}  /* end cu_sha1_update() */

/**
 * Finalize a SHA1 message digest.
 * Generate the SHA1 message digest and place in @a out.
 * @param ctx Pointer to SHA1 context
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_sha1_final(SHA1_CTX *ctx, void *out)
{
   uint32_t i, old;

   i = ctx->datalen;

   /* Pad whatever data is left in the buffer. */
   if (ctx->datalen < 56) {
      ctx->data[i++] = 0x80;
      memset(ctx->data + i, 0, 64 - i);
   } else if (ctx->datalen >= 56) {
      ctx->data[i++] = 0x80;
      if (i < 64) memset(ctx->data + i, 0, 64 - i);
      cu_sha1_transform(ctx, ctx->data);
      memset(ctx->data, 0, 56);
   }

   /* Append to the padding the total message's length in bits and
    * transform (big endian). */
   old = ctx->bitlen[0];
   ctx->bitlen[0] += (uint32_t) ctx->datalen << 3;
   if(ctx->bitlen[0] < old) ctx->bitlen[1]++;  /* add in carry */
   /* immitate bswap64() for bitlen */
   ((uint32_t *) ctx->data)[15] = bswap32(ctx->bitlen[0]);
   ((uint32_t *) ctx->data)[14] = bswap32(ctx->bitlen[1]);
   cu_sha1_transform(ctx, ctx->data);

   /* Since this implementation uses little endian byte ordering and
    * SHA uses big endian, reverse all the bytes when copying the
    * final state to the output hash. */
   ((uint32_t *) out)[0] = bswap32(ctx->state[0]);
   ((uint32_t *) out)[1] = bswap32(ctx->state[1]);
   ((uint32_t *) out)[2] = bswap32(ctx->state[2]);
   ((uint32_t *) out)[3] = bswap32(ctx->state[3]);
   ((uint32_t *) out)[4] = bswap32(ctx->state[4]);
}  /* end cu_sha1_final() */

/**
 * Convenient all-in-one SHA1 computation.
 * Performs cu_sha1_init(), cu_sha1_update() and cu_sha1_final(),
 * and places the resulting hash in @a out.
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_sha1(const void *in, size_t inlen, void *out)
{
   SHA1_CTX ctx;

   cu_sha1_init(&ctx);
   cu_sha1_update(&ctx, in, inlen);
   cu_sha1_final(&ctx, out);
}  /* end cu_sha1() */

/* CUDA kernel function */
__global__ static void kcu_sha1(const void *d_in, size_t *d_inlen,
   size_t max_inlen, void *d_out, int num)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid >= num) return;

   uint8_t *in = ((uint8_t *) d_in) + (tid * max_inlen);
   uint8_t *out = ((uint8_t *) d_out) + (tid * SHA1LEN);

   cu_sha1(in, d_inlen[tid], out);
}  /* end kcu_sha1() */

/* CUDA kernel testing function */
void test_kcu_sha1(const void *in, size_t *inlen, size_t max_inlen,
   void *out, int num)
{
   uint8_t *d_in, *d_out;
   size_t *d_inlen;

   hipMalloc(&d_in, num * max_inlen);
   hipMalloc(&d_inlen, num * sizeof(size_t));
   hipMalloc(&d_out, num * SHA1LEN);

   hipMemcpy(d_in, in, num * max_inlen, hipMemcpyHostToDevice);
   hipMemcpy(d_inlen, inlen, num * sizeof(size_t), hipMemcpyHostToDevice);
   hipMemset(d_out, 0, num * SHA1LEN);

   kcu_sha1<<<1, num>>>(d_in, d_inlen, max_inlen, d_out, num);

   hipMemcpy(out, d_out, num * SHA1LEN, hipMemcpyDeviceToHost);
}  /* end test_kcu_sha1() */

/* end include guard */
#endif
