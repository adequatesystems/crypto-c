#include "hip/hip_runtime.h"
/**
 * @private
 * @headerfile blake2b.cuh <blake2b.cuh>
 * @copyright This file is released into the Public Domain under
 * the Creative Commons Zero v1.0 Universal license.
*/

/* include guard */
#ifndef CRYPTO_BLAKE2B_CU
#define CRYPTO_BLAKE2B_CU


#include "blake2b.cuh"
#include <string.h>  /* for memory handling */

/**
 * Blake2b compression rounds.
 * @param ctx Pointer to Blake2b context
 * @param last Flag indicating the final compression
*/
__device__ void cu_blake2b_compress(BLAKE2B_CTX *ctx, int last)
{
   /* Blake2b compression Sigma */
   __constant__ static uint8_t Sigma[12][16] = {
      { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
      { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
      { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
      { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
      { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
      { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
      { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
      { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
      { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
      { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
      { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
      { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 }
   };
   uint64_t v[16];

   blake2b_compress_init(v, ctx->h, ctx->t, last);
   blake2b_compress_rounds(v, ctx->in.q, Sigma);
   blake2b_compress_set(v, ctx->h);
}  /* end cu_blake2b_compress() */

/**
 * Add @a inlen bytes from @a in to a Blake2b context for hashing.
 * @param ctx Pointer to Blake2b context
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
*/
__device__ void cu_blake2b_update(BLAKE2B_CTX *ctx, const void *in,
   size_t inlen)
{
   size_t i, n;

   for(i = n = 0; inlen; i += n, inlen -= n) {
      if (ctx->c == 128) {
         ctx->t[0] += ctx->c;
         if (ctx->t[0] < ctx->c) ctx->t[1]++;
         cu_blake2b_compress(ctx, 0);
         ctx->c = 0;
      }
      /* copy memory in chunks */
      n = (ctx->c + inlen) > 128 ? 128 - ctx->c : inlen;
      memcpy(ctx->in.b + ctx->c, (const uint8_t *) in + i, n);
      ctx->c += n;
   }
}  /* end cu_blake2b_update() */

/**
 * Initialize a Blake2b context with optional @a key.
 * To hash without a key, specify `NULL` and `0` for @a key and
 * @a keylen, respectively.
 * @param ctx Pointer to Blake2b context
 * @param key Pointer to optional "key" input
 * @param keylen Length of optional @a key input, in bytes
 * @param outlen Byte length of desired digest
 * @returns 0 on success, else if initialization fails (-1).
 * @note Blake2b initialization can fail if @a keylen is greater
 * than 64 or outlen is not a supported digest length. Supported
 * lengths include: 32 (256-bits), 48 (384-bits) or 64 (512-bits).
*/
__device__ int cu_blake2b_init(BLAKE2B_CTX *ctx, const void *key,
   int keylen, int outlen)
{
   if (keylen > 64) return -1;
   if (outlen != 32 && outlen != 48 && outlen != 64) return -1;

   ctx->c = 0;
   ctx->t[0] = 0;
   ctx->t[1] = 0;
   ctx->outlen = (uint64_t) outlen;
   ctx->h[0] = 0x6A09E667F3BCC908ull/* Blake2b_iv[0] */ ^
      0x01010000 ^ (keylen << 8) ^ outlen;
   ctx->h[1] = 0xBB67AE8584CAA73Bull/* Blake2b_iv[1] */;
   ctx->h[2] = 0x3C6EF372FE94F82Bull/* Blake2b_iv[2] */;
   ctx->h[3] = 0xA54FF53A5F1D36F1ull/* Blake2b_iv[3] */;
   ctx->h[4] = 0x510E527FADE682D1ull/* Blake2b_iv[4] */;
   ctx->h[5] = 0x9B05688C2B3E6C1Full/* Blake2b_iv[5] */;
   ctx->h[6] = 0x1F83D9ABFB41BD6Bull/* Blake2b_iv[6] */;
   ctx->h[7] = 0x5BE0CD19137E2179ull/* Blake2b_iv[7] */;

   /* zero remaining input buffer */
   memset(&ctx->in.b[keylen], 0, 128 - keylen);

   if (keylen > 0) {
      cu_blake2b_update(ctx, key, keylen);
      ctx->c = 128;
   }

   return 0;
}  /* end cu_blake2b_init() */

/**
 * Finalize a Blake2b message digest.
 * Generate the Blake2b message digest and place in @a out.
 * @param ctx Pointer to Blake2b context
 * @param out Pointer to location to place the message digest
*/
__device__ void cu_blake2b_final(BLAKE2B_CTX *ctx, void *out)
{
   ctx->t[0] += ctx->c;
   if (ctx->t[0] < ctx->c) ctx->t[1]++;

   /* zero remainder of input buffer */
   if (ctx->c < 128) memset(&ctx->in.b[ctx->c], 0, 128 - ctx->c);

   /* final compression */
   cu_blake2b_compress(ctx, 1);

   /* copy digest to out */
   memcpy(out, ctx->h, ctx->outlen);
}  /* end cu_blake2b_final() */

/**
 * Convenient all-in-one Blake2b computation.
 * Performs cu_blake2b_init(), cu_blake2b_update() and cu_blake2b_final(),
 * and places the resulting hash in @a out.
 * @param in Pointer to data to hash
 * @param inlen Length of @a in data, in bytes
 * @param key Pointer to optional "key" input
 * @param keylen Length of optional @a key input, in bytes
 * @param out Pointer to location to place the message digest
 * @param outlen Length* of desired message digest, in bytes<br/>
 * <sup>_*compatible message digest lengths are 32, 48 and 64_</sup>
*/
__device__ int cu_blake2b(const void *in, size_t inlen,
   const void *key, int keylen, void *out, int outlen)
{
   BLAKE2B_CTX ctx;

   if (cu_blake2b_init(&ctx, key, keylen, outlen)) {
      return -1;
   }
   cu_blake2b_update(&ctx, in, inlen);
   cu_blake2b_final(&ctx, out);

   return 0;
}  /* end cu_blake2b() */

/* CUDA kernel function */
__global__ static void kcu_blake2b(
   const void *d_in, size_t *d_inlen, size_t max_inlen,
   const void *d_key, int *d_keylen, int max_keylen,
   void *d_out, int outlen, int *d_ret, int num)
{
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid >= num) return;

   uint8_t *in = ((uint8_t *) d_in) + (tid * max_inlen);
   uint8_t *key = ((uint8_t *) d_key) + (tid * max_keylen);
   uint8_t *out = ((uint8_t *) d_out) + (tid * outlen);

   d_ret[tid] = cu_blake2b(in, d_inlen[tid], key, d_keylen[tid], out, outlen);
}  /* end kcu_blake2b() */

/* CUDA kernel testing function */
void test_kcu_blake2b(
   const void *in, size_t *inlen, size_t max_inlen,
   const void *key, int *keylen, int max_keylen,
   void *out, int outlen, int *ret, int num)
{
   uint8_t *d_in, *d_key, *d_out;
   size_t *d_inlen;
   int *d_keylen, *d_ret;

   hipMalloc(&d_ret, num * sizeof(int));
   hipMalloc(&d_in, num * max_inlen);
   hipMalloc(&d_inlen, num * sizeof(size_t));
   hipMalloc(&d_key, num * max_keylen);
   hipMalloc(&d_keylen, num * sizeof(int));
   hipMalloc(&d_out, num * outlen);

   hipMemset(d_ret, 0, num * sizeof(int));
   hipMemcpy(d_in, in, num * max_inlen, hipMemcpyHostToDevice);
   hipMemcpy(d_inlen, inlen, num * sizeof(size_t), hipMemcpyHostToDevice);
   hipMemcpy(d_key, key, num * max_keylen, hipMemcpyHostToDevice);
   hipMemcpy(d_keylen, keylen, num * sizeof(int), hipMemcpyHostToDevice);
   hipMemset(d_out, 0, num * outlen);

   kcu_blake2b<<<1, num>>>(
      d_in, d_inlen, max_inlen,
      d_key, d_keylen, max_keylen,
      d_out, outlen, d_ret, num);

   hipMemcpy(ret, d_ret, num * sizeof(int), hipMemcpyDeviceToHost);
   hipMemcpy(out, d_out, num * outlen, hipMemcpyDeviceToHost);
}  /* end test_kcu_blake2b() */

/* end include guard */
#endif
