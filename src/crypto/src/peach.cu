#include "hip/hip_runtime.h"
/*
 * cuda_trigg.cu  Multi-GPU CUDA Mining
 *
 * Copyright (c) 2019 by Adequate Systems, LLC.  All Rights Reserved.
 * See LICENSE.PDF   **** NO WARRANTY ****
 *
 * Date: 10 August 2018
 * Revision: 31
 */

#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <nvml.h>

#include "../../config.h"
#include "peach.h"
#include "nighthash.cu"
#include "cuda_peach.h"

__constant__ static uint8_t __align__(8) c_phash[32];
__constant__ static uint8_t __align__(8) c_input[108];
__constant__ static uint8_t __align__(8) c_difficulty;
__constant__ static int Z_MASS[4] = {238,239,240,242};
__constant__ static int Z_ING[2]  = {42,43};
__constant__ static int Z_TIME[16] =
   {82,83,84,85,86,87,88,243,249,250,251,252,253,254,255,253};
__constant__ static int Z_AMB[16] =
   {77,94,95,96,126,214,217,218,220,222,223,224,225,226,227,228};
__constant__ static int Z_ADJ[64] =
   {61,62,63,64,65,66,67,68,69,70,71,72,73,74,75,76,77,78,79,80,81,82,83,84,85,86,87,
    88,89,90,91,92,94,95,96,97,98,99,100,101,102,103,104,105,107,108,109,110,112,114,
    115,116,117,118,119,120,121,122,123,124,125,126,127,128};

inline int cudaCheckError( const char *msg, uint32_t gpu, const char *file)
{
   hipError_t err = hipGetLastError();
   if(hipSuccess != err) {
      fprintf(stderr, "%s Error (#%d) in %s: %s\n",
              msg, gpu, file, hipGetErrorString(err));
      return 1;
   }
   return 0;
}


__device__ uint32_t cuda_next_index(uint32_t index, uint8_t *g_map, uint8_t *nonce)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[HASHLEN + 4 + TILE_LENGTH];
   byte hash[HASHLEN];
   int i, seedlen;

   /* Create nighthash seed for this index on the map */
   seedlen = HASHLEN + 4 + TILE_LENGTH;
   memcpy(seed, nonce, HASHLEN);
   memcpy(seed + HASHLEN, (byte *) &index, 4);
   memcpy(seed + HASHLEN + 4, &g_map[index * TILE_LENGTH], TILE_LENGTH);
   
   /* Setup nighthash the seed, NO TRANSFORM */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 0);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, hash);

   /* Convert 32-byte Hash Value Into 8x 32-bit Unsigned Integer */
   for(i = 0, index = 0; i < 8; i++)
      index += ((uint32_t *) hash)[i];

   return index % MAP;
}


__device__ void cuda_gen_tile(uint32_t index, uint8_t *g_map)
{
   CUDA_NIGHTHASH_CTX nighthash;
   byte seed[4 + HASHLEN];
   byte *tilep;
   int i, j, seedlen;

   /* Set map pointer */
   tilep = &g_map[index * TILE_LENGTH];

   /* Create nighthash seed for this index on the map */
   seedlen = 4 + HASHLEN;
   memcpy(seed, (byte *) &index, 4);
   memcpy(seed + 4, c_phash, HASHLEN);

   /* Setup nighthash with a transform of the seed */
   cuda_nighthash_init(&nighthash, seed, seedlen, index, 1);

   /* Update nighthash with the seed data */
   cuda_nighthash_update(&nighthash, seed, seedlen);

   /* Finalize nighthash into the first 32 byte chunk of the tile */
   cuda_nighthash_final(&nighthash, tilep);

   /* Begin constructing the full tile */
   for(i = 0; i < TILE_LENGTH; i += HASHLEN) { /* For each tile row */
      /* Set next row's pointer location */
      j = i + HASHLEN;

      /* Hash the current row to the next, if not at the end */
      if(j < TILE_LENGTH) {
         /* Setup nighthash with a transform of the current row */
         cuda_nighthash_init(&nighthash, &tilep[i], HASHLEN, index, 1);

         /* Update nighthash with the seed data and tile index */
         cuda_nighthash_update(&nighthash, &tilep[i], HASHLEN);
         cuda_nighthash_update(&nighthash, (byte *) &index, 4);

         /* Finalize nighthash into the first 32 byte chunk of the tile */
         cuda_nighthash_final(&nighthash, &tilep[j]);
      }
   }
}


__global__ void cuda_build_map(uint8_t *g_map)
{
   const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;
   if (thread < MAP)
      cuda_gen_tile(thread, g_map);
}


__global__ void cuda_find_peach(uint32_t threads, uint8_t *g_map,
                                int32_t *g_found, uint8_t *g_seed)
{
   const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

   CUDA_SHA256_CTX ictx;
   uint8_t seed[16] = {0}, nonce[32] = {0};
   uint8_t bt_hash[32], fhash[32];
   int32_t i, j, n, x;
   uint32_t sm;

   if (thread < threads) {
      /* Determine second seed */
      if(thread < 131072) { /* This frame permutations: 131,072 */
         seed[ 0] = Z_TIME[(thread & 15)];
         seed[ 1] = Z_AMB[(thread >> 4) & 15];
         seed[ 2] = 1;
         seed[ 3] = Z_ADJ[(thread >> 8) & 63];
         seed[ 4] = Z_MASS[(thread >> 14) & 3];
         seed[ 5] = 1;
         seed[ 6] = Z_ING[(thread >> 16) & 1];
      }

      /* store full nonce */
      #pragma unroll
      for (i = 0; i < 16; i++)
         nonce[i] = c_input[i + 92];

      #pragma unroll
      for (i = 0; i < 16; i++)
         nonce[i+16] = seed[i];

      /*********************************************************/
      /* Hash 124 bytes of Block Trailer, including both seeds */

      cuda_sha256_init(&ictx);
      cuda_sha256_update(&ictx, c_input, 108);
      cuda_sha256_update(&ictx, seed, 16);
      cuda_sha256_final(&ictx, bt_hash);

      /****************************************************/
      /* Follow the tile path based on the selected nonce */
      
      sm = bt_hash[0];
      #pragma unroll
      for(i = 1; i < HASHLEN; i++)
         sm *= bt_hash[i];
      sm %= MAP;

      /* make <JUMP> tile jumps to find the final tile */
      #pragma unroll
      for(j = 0; j < JUMP; j++)
        sm = cuda_next_index(sm, g_map, nonce);

      /****************************************************************/
      /* Check the hash of the final tile produces the desired result */

      cuda_sha256_init(&ictx);
      cuda_sha256_update(&ictx, bt_hash, HASHLEN);
      cuda_sha256_update(&ictx, &g_map[sm * TILE_LENGTH], TILE_LENGTH);
      cuda_sha256_final(&ictx, fhash);

      /* Evaluate hash */
      for (x = i = j = n = 0; i < HASHLEN; i++) {
         x = fhash[i];
         if (x != 0) {
            for(j = 7; j > 0; j--) {
               x >>= 1;
               if(x == 0) {
                  n += j;
                  break;
               }
            }
            break;
         }
         n += 8;
      }

      if(n >= c_difficulty && !atomicExch(g_found, 1)) {
         /* PRINCESS FOUND! */
         #pragma unroll
         for (i = 0; i < 16; i++)
            g_seed[i] = seed[i];
      }
      /* Our princess is in another castle ! */
   }
}


extern "C" {


uint8_t enable_nvml = 0;
GPU_t gpus[MAX_GPUS] = { 0 };
uint32_t num_gpus = 0;
/* Max 63 GPUs Supported */
PeachCudaCTX peach_ctx[64];
PeachCudaCTX *ctx = peach_ctx;
dim3 grid(512);
dim3 block(256);
uint32_t threads = 131072;
int32_t nGPU = 0;
int32_t *found;
byte gpuInit = 0;
byte bnum[8] = {0};
byte *diff;
byte *phash;

int init_nvml() {
   int32_t num_cuda = 0;
   hipError_t cr = hipGetDeviceCount(&num_cuda);
   if (num_cuda > MAX_GPUS) num_cuda = MAX_GPUS;

   for (int i = 0; i < num_cuda; i++) {
      struct hipDeviceProp_t p = { 0 };
      hipError_t cr = hipGetDeviceProperties(&p, i);
      printf("CUDA pciDomainID: %x, pciBusID: %x, pciDeviceID: %x\n", p.pciDomainID, p.pciBusID, p.pciDeviceID);
      gpus[i].pciDomainId = p.pciDomainID;
      gpus[i].pciBusId = p.pciBusID;
      gpus[i].pciDeviceId = p.pciDeviceID;
      gpus[i].cudaNum = i;
      num_gpus++;
   }


   nvmlReturn_t r = nvmlInit();
   if (r != NVML_SUCCESS) {
      printf("Failed to initialize NVML: %s\n", nvmlErrorString(r));
      enable_nvml = 0;
      return 0;
   }
   uint32_t nvml_device_count;
   r = nvmlDeviceGetCount(&nvml_device_count);
   if (r != NVML_SUCCESS) {
      printf("Failed to get NVML device count: %s\n", nvmlErrorString(r));
      enable_nvml = 0;
      return 0;
   }
   printf("NVML Devices: %d\n", nvml_device_count);
   for (int i = 0; i < nvml_device_count; i++) {
      nvmlDevice_t dev;
      r = nvmlDeviceGetHandleByIndex(i, &dev);
      if (r != NVML_SUCCESS) {
         printf("nvmlDeviceGetHandleByIndex failed: %s\n", nvmlErrorString(r));
         nvml_device_count = i;
         break;
      }
      nvmlPciInfo_t pci;
      r = nvmlDeviceGetPciInfo(dev, &pci);
      if (r != NVML_SUCCESS) {
         printf("nvmlDeviceGetPciInfo failed: %s\n", nvmlErrorString(r));
         continue;
      }
      printf("NVML PCI: pciDeviceId: %x, pciSubSystemId: %x, domain: %x, device: %x, bus: %x\n", pci.pciDeviceId, pci.pciSubSystemId, pci.domain, pci.device, pci.bus);

      for (int j = 0; j < num_cuda; j++) {
         if (gpus[j].pciDomainId == pci.domain && gpus[j].pciBusId == pci.bus && gpus[i].pciDeviceId == pci.device) {
            printf("NVML device is CUDA Device: %d\n", gpus[j].cudaNum);
            gpus[j].nvml_dev = dev;
            break;
         }
      }

      char device_name[128];
      r = nvmlDeviceGetName(dev, device_name, 128);
      if (r != NVML_SUCCESS) {
         printf("nvmlDeviceGetName failed: %s\n", nvmlErrorString(r));
      }
      else {
         printf("Device: %d, Name: %s\n", i, device_name);
      }
   }
   enable_nvml = 1;
   return 1;
}

int init_cuda_peach(byte difficulty, byte *prevhash, byte *blocknumber) {
   int i;
   
   /* Obtain and check system GPU count */
   nGPU = 0;
   hipGetDeviceCount(&nGPU);
   if(nGPU<1 || nGPU>64) return nGPU;
   
   /* Allocate pinned host memory */
   hipHostMalloc(&found, 4);
   hipHostMalloc(&diff, 1);
   hipHostMalloc(&phash, 32);
   
   /* Copy immediate block data to pinned memory */
   *found = 0;
   *diff = difficulty;
   memcpy(phash, prevhash, 32);
   
   /* Initialize GPU context init variable incase
    * it holds a random number from memory */
   if(gpuInit == 0) {
      gpuInit = 1;
      for (i = 0; i < nGPU; i++)
         ctx[i].init = 0;
   }
   
   /* Initialize GPU data asynchronously */
   for (i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      
      /* Create Stream */
      hipStreamCreate(&ctx[i].stream);
      
      /* Allocate device memory */
      hipMalloc(&ctx[i].d_found, 4);
      hipMalloc(&ctx[i].d_seed, 16);
      
      /* Allocate associated device-host memory */
      hipHostMalloc(&ctx[i].seed, 16);
      hipHostMalloc(&ctx[i].input, 108);
      
      /* Copy immediate block data to device memory */
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_difficulty), diff, 1, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
      hipMemcpyToSymbolAsync(HIP_SYMBOL(c_phash), phash, 32, 0,
                              hipMemcpyHostToDevice, ctx[i].stream);
      
      /* Set remaining device memory */
      hipMemsetAsync(ctx[i].d_found, 0, 4, ctx[i].stream);
      hipMemsetAsync(ctx[i].d_seed, 0, 16, ctx[i].stream);
      
      /* Set initial round variables */
      ctx[i].next_seed[0] = 0;
      
      /* If first init, setup map and cache */
      if(ctx[i].init == 0) {
         /* NOTE: The device MAP that holds the data of a map DOES NOT
          * explicitly get free()'d. The reason behind this is because
          * we reuse the map variable between blocks, and just rebuild
          * the map once every block. The GPU free's the MAP when the
          * program ends by default. This can be adjusted later. */
         hipMalloc(&ctx[i].d_map, MAP_LENGTH);
         ctx[i].init = 1;
      }
      
      /* (re)Build map if new block */
      if(memcmp(bnum, blocknumber, 8) != 0)
         cuda_build_map<<<4096, 256, 0, ctx[i].stream>>>(ctx[i].d_map);
   }
   
   /* Check for any GPU initialization errors */
   for(i = 0; i < nGPU; i++) {
      hipSetDevice(i);
      hipStreamSynchronize(ctx[i].stream);
      if(cudaCheckError("init_cuda_peach()", i, __FILE__))
         return -1;
   }
   
   /* Update block number */
   memcpy(bnum, blocknumber, 8);

   return nGPU;
}

void free_cuda_peach() {
   int i;
   
   /* Free pinned host memory */
   hipHostFree(diff);
   hipHostFree(found);
   hipHostFree(phash);
   
   /* Free GPU data */
   for (i = 0; i<nGPU; i++) {
      hipSetDevice(i);
      
      /* Destroy Stream */
      hipStreamDestroy(ctx[i].stream);
      
      /* Free device memory */
      hipFree(ctx[i].d_found);
      hipFree(ctx[i].d_seed);
      hipFree(ctx[i].d_map);
      
      /* Free associated device-host memory */
      hipHostFree(ctx[i].seed);
      hipHostFree(ctx[i].input);
   }
}

extern byte *trigg_gen(byte *in);

__host__ void cuda_peach(byte *bt, uint32_t *hps, byte *runflag)
{
   int i, j;
   uint64_t lastnHaiku, nHaiku = 0;
   time_t seconds = time(NULL);
   time_t gpu_stats_time = time(NULL);
   for( ; *runflag && *found == 0; ) {
      for (i=0; i<nGPU; i++) {
         /* Prepare next seed for GPU... */
         if(ctx[i].next_seed[0] == 0) {
            /* ... generate first GPU seed (and expand as Haiku) */
            trigg_gen(ctx[i].next_seed);

            /* ... and prepare round data */
            memcpy(ctx[i].input, bt, 92);
            memcpy(ctx[i].input+92, ctx[i].next_seed, 16);
         }
         /* Check if GPU has finished */
         hipSetDevice(i);
         if(hipStreamQuery(ctx[i].stream) == hipSuccess) {
            hipMemcpy(found, ctx[i].d_found, 4, hipMemcpyDeviceToHost);
            if(*found==1) { /* SOLVED A BLOCK! */
               hipMemcpy(ctx[i].seed, ctx[i].d_seed, 16, hipMemcpyDeviceToHost);
               memcpy(bt + 92, ctx[i].curr_seed, 16);
               memcpy(bt + 92 + 16, ctx[i].seed, 16);
               break;
            }
            // Calculate per GPU HPS
            gettimeofday(&(ctx[i].t_end), NULL);
            uint64_t ustart = 1000000 * ctx[i].t_start.tv_sec + ctx[i].t_start.tv_usec;
            if (ustart > 0) {
               uint64_t uend = 1000000 * ctx[i].t_end.tv_sec + ctx[i].t_end.tv_usec;
               double tdiff = (uend - ustart) / 1000.0 / 1000.0;
               ctx[i].hps_index = (ctx[i].hps_index + 1) % 3;
               ctx[i].hps[ctx[i].hps_index] = threads / tdiff;
               uint32_t shps = 0;
               for (int j = 0; j < 3; j++) {
                  shps += ctx[i].hps[j];
               }
               ctx[i].ahps = shps / 3;
            }
            // End per GPU HPS

            gettimeofday(&(ctx[i].t_start), NULL);

            /* Send new GPU round Data */
            hipMemcpyToSymbolAsync(HIP_SYMBOL(c_input), ctx[i].input, 108, 0,
                                    hipMemcpyHostToDevice, ctx[i].stream);
            /* Start GPU round */
            cuda_find_peach<<<grid, block, 0, ctx[i].stream>>>(threads,
                                 ctx[i].d_map, ctx[i].d_found, ctx[i].d_seed);

            /* Add to haiku count */
            nHaiku += threads;

            /* Store round vars aside for checks next loop */
            memcpy(ctx[i].curr_seed,ctx[i].next_seed,16);
            ctx[i].next_seed[0] = 0;
         }
         
         /* Waiting on GPU? ... */
         if(cudaCheckError("cuda_peach()", i, __FILE__)) {
            *runflag = 0;
            return;
         }
      }

      // Print GPU stats every 5 seconds
      if ( (time(NULL) - gpu_stats_time) > 5 ) {
         for (j = 0; j < nGPU; j++) {
            if (enable_nvml) {
               uint32_t temp = 0;
               uint32_t power = 0;
               nvmlReturn_t r = nvmlDeviceGetTemperature(gpus[j].nvml_dev, NVML_TEMPERATURE_GPU, &temp);
               if (r != NVML_SUCCESS) {
                  printf("nvmlDeviceGetTemperature failed: %s\n", nvmlErrorString(r));
               }

               r = nvmlDeviceGetPowerUsage(gpus[j].nvml_dev, &power);
               if (r != NVML_SUCCESS) {
                  printf("nvmlDeviceGetPowerUsage Failed: %s\n", nvmlErrorString(r));
               }
               gpus[j].temp = temp;
               gpus[j].power = power;

               printf("GPU %d: %7d H/s, Temperature: %d C, Power: %6.2f W\n", j,
                     ctx[j].ahps, gpus[j].temp, gpus[j].power / 1000.0);
            } else {
               printf("GPU %d: %7d H/s\n", j, ctx[j].ahps);
            }
         }
         gpu_stats_time = time(NULL);
      }
      
      /* Chill a bit if nothing is happening */
      if(lastnHaiku == nHaiku) usleep(1000);
      else lastnHaiku = nHaiku;
   }
    
   seconds = time(NULL) - seconds;
   if(seconds == 0) seconds = 1;
   nHaiku /= seconds;
   *hps = (uint32_t) nHaiku;
}


}
